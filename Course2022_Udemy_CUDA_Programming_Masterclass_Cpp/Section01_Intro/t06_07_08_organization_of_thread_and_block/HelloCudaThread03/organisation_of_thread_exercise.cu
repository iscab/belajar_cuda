#include "hip/hip_runtime.h"
﻿
/**
* Organization of thread and block (exercise): cu file
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why
* notes:  use CUDA option in New Project in Visual C++ 2022
*
* version: 09:25 15.11.2022
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_details()
{
	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d,  blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d  \n",
		threadIdx.x, threadIdx.y, threadIdx.z,
		blockIdx.x, blockIdx.y, blockIdx.z,
		blockDim.x, blockDim.y,
		gridDim.x, gridDim.y);
}


int main()
{
	int nx, ny, nz;
	// 3D grid which has 4 threads in all X, Y, Z dimension  
	nx = 4;
	ny = 4;
	nz = 4;

	// thread block size will be 2 threads in each dimension  
	dim3 block(2, 2, 2);
	dim3 grid(nx/block.x, ny/block.y, nz/block.z);

	print_details << < grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

/**
* version: 09:25 15.11.2022
*
* End of file
*/
