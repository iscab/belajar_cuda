#include "hip/hip_runtime.h"
﻿/**
* Unique index calculation for thread in a grid: cu file
* (threadIdx, blockIdx, blockDim)
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why
* notes:  use CUDA option in New Project in Visual C++ 2022
*
* version: 10:45 17.11.2022
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void unique_idx_calc_threadIdx(int* input)
{
	int tid = threadIdx.x;
	printf("threadIdx : %d,  value : %d \n", tid, input[tid]);
}

int main()
{
	// array of data of the host
	int array_size = 8;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23,9,4,53,65,12,1,33 };

	for (int ii = 9; ii < array_size; ii++)
	{
		printf("%d", h_data[ii]);
	}
	printf("\n \n");

	// array of data of CUDA device  
	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	// threadblocks in a grid
	dim3 block(8);
	dim3 grid(1);

	unique_idx_calc_threadIdx << < grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

/**
* version: 10:45 17.11.2022
*
* End of file
*/
