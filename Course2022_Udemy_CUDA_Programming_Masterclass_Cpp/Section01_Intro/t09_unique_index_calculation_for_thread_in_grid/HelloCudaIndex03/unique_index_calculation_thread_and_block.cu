#include "hip/hip_runtime.h"
﻿/**
* Unique index calculation for thread in a grid: cu file
* (threadIdx, blockIdx, blockDim)
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why
* notes:  use CUDA option in New Project in Visual C++ 2022
*
* version: 09:05 18.11.2022
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void unique_idx_calc_threadIdx(int* input)
{
	int tid = threadIdx.x;
	printf("threadIdx : %d,  value : %d \n", tid, input[tid]);
}

__global__ void unique_gid_calculation(int* input)
{
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x : %d, threadIdx.x : %d, gid : %d, value : %d \n",
		blockIdx.x, tid, gid, input[gid]);
}

int main()
{
	// array of data of the host
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23,9,4,53,65,12,1,33, 87,45,23,12,342,56,44,99 };

	for (int ii = 0; ii < array_size; ii++)
	{
		printf("%d ", h_data[ii]);
	}
	printf("\n \n");

	// array of data of CUDA device  
	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	// threadblocks in a grid
	dim3 block(4);
	dim3 grid(4);
	// 4 threadblocks in a grid, each threadblock has 4 threads  

	unique_gid_calculation << < grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

/**
* version: 09:05 18.11.2022
*
* End of file
*/
