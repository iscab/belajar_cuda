#include "hip/hip_runtime.h"
/**
* Hello CUDA World cu file
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* - a main function in a cpp file, including a cuh header file
* note: this is a work-around to compile cu files. 
*
* version: 10:51 29.07.2022
*/

#include "hip/hip_runtime.h"
#include ""
#include "./hello_cuda.cuh"

#include <stdio.h>

__global__ void hello_cuda_kernel()
{
	printf("Hello CUDA World \n");
}

/* Wrapper function
*/
int hello_cuda()
{
	// Example 1: 
	//hello_cuda_kernel << < 1, 1 >> > ();
	//hipDeviceSynchronize();

	// Example 2:
	//hello_cuda_kernel << < 1, 20 >> > ();
	//hipDeviceSynchronize();

	// Example 3:
	//dim3 block(4);
	//dim3 grid(8);
	//hello_cuda_kernel << < grid, block >> > ();
	//hipDeviceSynchronize();

	// Example 4:
	//dim3 block(8,2);
	//dim3 grid(2,2);
	//hello_cuda_kernel << < grid, block >> > ();
	//hipDeviceSynchronize();

	// Example 5:
	int nx, ny;
	nx = 16;
	ny = 4;
	dim3 block(8, 2);
	dim3 grid(nx / block.x, ny / block.y);
	hello_cuda_kernel << < grid, block >> > ();
	hipDeviceSynchronize();


	hipDeviceReset();
	return 0;
}

/**
* version: 10:51 29.07.2022
*
* End of file
*/
