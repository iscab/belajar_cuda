#include "hip/hip_runtime.h"
﻿/**
* Hello CUDA World
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why  
* notes:  use CUDA option in New Project in Visual C++ 2022  
* 
* 
* version: 10:55 09.11.2022
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello CUDA World \n");
}

int main()
{
	hello_cuda << < 1, 1 >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

/**
* version: 10:55 09.11.2022
*
* End of file
*/
