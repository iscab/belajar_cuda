#include "hip/hip_runtime.h"
﻿/**
* Unique index calculation for thread in a 2D 2D grid: cu file
* (threadIdx, blockIdx, blockDim)
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why
* notes:  use CUDA option in New Project in Visual C++ 2022
*
* version: 09:40 06.02.2023
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void unique_gid_calculation_2d_2d(int* input)
{
	int tid = blockDim.x * threadIdx.y + threadIdx.x;

	int num_threads_in_a_block = blockDim.x * blockDim.y;
	int block_offset = blockIdx.x * num_threads_in_a_block;

	int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
	int row_offset = num_threads_in_a_row * blockIdx.y;

	int gid = tid + block_offset + row_offset;
	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, threadIdx.y : %d, tid : %d, gid : %d, value : %d \n",
		blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, tid, gid, input[gid]);
}

int main()
{
	// array of data of the host
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23,9,4,53,65,12,1,33,22,43,56,4,76,81,94,32 };

	for (int ii = 0; ii < array_size; ii++)
	{
		printf("%d ", h_data[ii]);
	}
	printf("\n \n");

	// array of data of CUDA device  
	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	// threadblocks in a grid
	dim3 block(2, 2);
	dim3 grid(2, 2);

	unique_gid_calculation_2d_2d << < grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

/**
* version: 09:40 06.02.2023
*
* End of file
*/
