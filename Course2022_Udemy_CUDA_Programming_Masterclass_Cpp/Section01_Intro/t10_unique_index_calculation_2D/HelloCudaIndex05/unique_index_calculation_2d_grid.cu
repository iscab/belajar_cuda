#include "hip/hip_runtime.h"
﻿/**
* Unique index calculation for thread in a 2D grid: cu file
* (threadIdx, blockIdx, blockDim)
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why
* notes:  use CUDA option in New Project in Visual C++ 2022
*
* version: 10:37 01.02.2023
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void unique_gid_calculation_2d(int* input)
{
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d, value : %d \n",
		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

int main()
{
	// array of data of the host
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23,9,4,53,65,12,1,33,22,43,56,4,76,81,94,32 };

	for (int ii = 0; ii < array_size; ii++)
	{
		printf("%d ", h_data[ii]);
	}
	printf("\n \n");

	// array of data of CUDA device  
	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	// threadblocks in a grid
	dim3 block(4);
	dim3 grid(2,2);

	unique_gid_calculation_2d << < grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

/**
* version: 10:37 01.02.2023
*
* End of file
*/
