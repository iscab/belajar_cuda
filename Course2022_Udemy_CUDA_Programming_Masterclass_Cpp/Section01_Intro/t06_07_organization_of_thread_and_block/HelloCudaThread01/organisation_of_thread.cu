#include "hip/hip_runtime.h"
/**
* Organization of thread and block: cu file
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* - a main function in a cpp file, including a cuh header file
* note: this is a work-around to compile cu files.
*
* version: 08:21 11.08.2022
*/

#include "hip/hip_runtime.h"
#include ""
#include "./organisation_of_thread.cuh"

#include <stdio.h>

__global__ void print_threadIds()
{
	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d  \n",
		threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_details()
{
	printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d  \n",
		blockIdx.x, blockIdx.y, blockIdx.z, 
		blockDim.x, blockDim.y,
		gridDim.x, gridDim.y);
}

int organisation_of_thread()
{
	int nx, ny;
	nx = 16;
	ny = 16;

	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);

	// Example 1: threadId
	//print_threadIds << < grid, block >> > ();
	//hipDeviceSynchronize();

	// Example 2: blockId
	print_details << < grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

/**
* version: 08:21 11.08.2022
*
* End of file
*/
