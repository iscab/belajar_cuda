#include "hip/hip_runtime.h"
﻿/**
* Organization of thread and block: cu file
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why  
* notes:  use CUDA option in New Project in Visual C++ 2022  
*
* version: 09:45 11.11.2022
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_threadIds()
{
	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d  \n",
		threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_details()
{
	printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d  \n",
		blockIdx.x, blockIdx.y, blockIdx.z,
		blockDim.x, blockDim.y,
		gridDim.x, gridDim.y);
}

__global__ void print_more_details()
{
	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d,  blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d  \n",
		threadIdx.x, threadIdx.y, threadIdx.z,
		blockIdx.x, blockIdx.y, blockIdx.z,
		blockDim.x, blockDim.y,
		gridDim.x, gridDim.y);
}

int main()
{
	int nx, ny;
	nx = 16;
	ny = 16;

	// 8 by 8 threadblocks, so a block has 8 by 8 threads. A grid will have 2 by 2 blocks.
	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);

	// Example 1: threadId
	//print_threadIds << < grid, block >> > ();
	//hipDeviceSynchronize();

	// Example 2: blockId
	//print_details << < grid, block >> > ();
	//hipDeviceSynchronize();

	// Example 3: threadId & blockId
	print_more_details << < grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}


/**
* version: 09:45 11.11.2022
*
* End of file
*/
