#include "hip/hip_runtime.h"
﻿/**
* Memory tranfer from host to device: cu file
* when the host data is smaller than device memory
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why
* notes:  use CUDA option in New Project in Visual C++ 2022
*
* version: 09:42 06.02.2023
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include<cstring>
#include<time.h>

__global__ void mem_trs_test(int* input)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("tid : %d, gid: %d, value : %d \n", threadIdx.x, gid, input[gid]);
}

__global__ void mem_trs_test2(int* input, int size)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		printf("tid : %d, gid: %d, value : %d \n", threadIdx.x, gid, input[gid]);
	}		
	else
	{
		printf("tid : %d, gid: %d, value : dodol \n", threadIdx.x, gid);
	}
}

int main()
{
	// size of the array
	int size = 150;
	int byte_size = size * sizeof(int);

	// array of data of the host
	int* h_input;
	h_input = (int*)malloc(byte_size);

	// random numbers in the array
	time_t t;
	srand((unsigned)time(&t));  // seeding, before randomization  
	for (int ii = 0; ii < size; ii++)
	{
		h_input[ii] = (int)(rand() & 0xff);
	}

	// array of data of the device
	int* d_input;
	hipMalloc((void**)&d_input, byte_size);

	// transfer memory from host to device
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

	// threadblocks in a grid
	dim3 block(32);
	dim3 grid(5);

	mem_trs_test2 << < grid, block >> > (d_input, size);
	hipDeviceSynchronize();

	// free the memory
	hipFree(d_input);
	free(h_input);

	hipDeviceReset();
	return 0;
}

/**
* version: 09:42 06.02.2023
*
* End of file
*/
