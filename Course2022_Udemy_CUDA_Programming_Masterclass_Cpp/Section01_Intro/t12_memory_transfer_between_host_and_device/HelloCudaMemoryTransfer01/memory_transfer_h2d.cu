#include "hip/hip_runtime.h"
﻿/**
* Memory transfer from host to device: cu file
*
* it is working, after compiled with
* - Visual C++ 2022
* - CUDA 11.7
* no compiling error, but I don't know why
* notes:  use CUDA option in New Project in Visual C++ 2022
*
* version: 09:43 06.02.2023
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include<cstring>
#include<time.h>

__global__ void mem_trs_test(int* input)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("tid : %d, gid: %d, value : %d \n", threadIdx.x, gid, input[gid]);
}

int main()
{
	// size of the array
	int size = 128;
	int byte_size = size * sizeof(int);

	// array of data of the host
	int* h_input;
	h_input = (int*)malloc(byte_size);

	// random numbers in the array
	time_t t;
	srand((unsigned)time(&t));  // seeding, before randomization  
	for (int ii = 0; ii < size; ii++)
	{
		h_input[ii] = (int)(rand()&0xff);
	}

	// array of data of the device
	int* d_input;
	hipMalloc((void**)&d_input,byte_size);

	// transfer memory from host to device
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

	// threadblocks in a grid
	dim3 block(64);
	dim3 grid(2);

	mem_trs_test << < grid, block >> > (d_input);
	hipDeviceSynchronize();

	// free the memory
	hipFree(d_input);
	free(h_input);

	hipDeviceReset();
	return 0;
}

/**
* version: 09:43 06.02.2023
*
* End of file
*/
